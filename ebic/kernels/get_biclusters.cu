#include "hip/hip_runtime.h"
/***

Copyright (c) 2017 Patryk Orzechowski

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

***/


#ifndef _GET_BICLUSTERS_CU_
#define _GET_BICLUSTERS_CU_

#include "evaluate_trends.cu"

template <typename T>
__global__ void get_biclusters(const int SHARED_MEM_SIZE,
                        const float APPROX_TRENDS_RATIO,
                        const int NEGATIVE_TRENDS_ENABLED,
                        const float EPSILON,
                        int num_biclusters,
                        int *bicl_indices,
                        int size_indices,
                        int *compressed_biclusters,
                        int num_rows,
                        int num_cols,
                        T *data,
                        int *coverage) {
  extern __shared__ int memory[];
  int *trend_increasing=memory;
  int *trend_decreasing=&trend_increasing[SHARED_MEM_SIZE];
  //float *trendvalue=(float*)&trend_decreasing[SHARED_MEM_SIZE];
  T *trendvalue=(T*)&trend_decreasing[SHARED_MEM_SIZE];

  long long int index_x = blockIdx.x * blockDim.x + threadIdx.x;    //block of bicluster
  long long int index_y = blockIdx.y * blockDim.y + threadIdx.y;    //block of row


  evaluate_trends<T>(bicl_indices, compressed_biclusters, num_rows, num_cols, data, trend_increasing, trendvalue, EPSILON);



  if (trend_increasing[threadIdx.y]<APPROX_TRENDS_RATIO*(bicl_indices[index_x+1]-bicl_indices[index_x])) {
    trend_increasing[threadIdx.y]=0;
  } else
    trend_increasing[threadIdx.y]=1;

  if (NEGATIVE_TRENDS_ENABLED) {
    evaluate_trends<T>(bicl_indices, compressed_biclusters, num_rows, num_cols, data, trend_decreasing, trendvalue, EPSILON, -1);
    if (trend_decreasing[threadIdx.y]<APPROX_TRENDS_RATIO*(bicl_indices[index_x+1]-bicl_indices[index_x])) {
      trend_decreasing[threadIdx.y]=0;
    } else {
      trend_decreasing[threadIdx.y]=1;
    }
  }
  __syncthreads();

  if (index_y<num_rows && index_x<num_biclusters) {
    coverage[index_y+num_rows*index_x]=trend_increasing[threadIdx.y]|trend_decreasing[threadIdx.y];
  }
}


#endif